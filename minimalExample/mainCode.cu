#include "hip/hip_runtime.h"

//    gcc importedCFile.c mainCode.c -O3 -lm -Wall -o mainCode -lrt
//    nvcc -lm -Xcompiler -Wall importedCFile.c mainCode.cu -o mainCode

// If lines 10 to 13 and 54 are commented it works ok


#include <stdio.h>
#include <stdlib.h>
extern "C" {
	#include "importedCFile.h"
}

int *n, *device_n;                 

/**
* Return a dynamic allocated vector of n integer elements
**/
int *getVector(int n)  {
    int *Ptr = (int *)malloc(n * sizeof(int));
    if (Ptr == NULL) { fprintf(stderr,"Memory error in getVector!\n"); exit(-1); }
    return Ptr;
}

// A simple kernel
__global__
void aKernel(int n, int a, int *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) x[i] += i ;
}

/**
* Main Program
**/
int main(int argc, char *argv[])  
{   int N = 256;
	
	n = getVector(N);								//	Host array
    
	hipMalloc(&device_n, N*sizeof(int)); 			//	Device arrays

	for (int i = 0; i < N; i++) {					//	Initialize array
	    n[i] = 1;
	}

	hipMemcpy(device_n, n, N*sizeof(int), hipMemcpyHostToDevice);		//	Copy host array to device array

	///KERNEL_NAME <<< N_BLOCKS, N_THREAD_PER_BLOCK >>> PARAMS 
	aKernel<<<1, 256>>>(N, 2, device_n);

	hipMemcpy(n, device_n, N*sizeof(int), hipMemcpyDeviceToHost);		//	Copy device array to host array

    anExample();

	for (int i = 0; i < N; i++) {					//	Initialize array
	   printf("%d\n", n[i] );
	}

	hipFree(device_n);
	free(n);    

    return 0;

}